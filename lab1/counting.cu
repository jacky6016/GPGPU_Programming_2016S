#include "hip/hip_runtime.h"
#include "counting.h"
#include <cstdio>
#include <cassert>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <thrust/functional.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>

__device__ __host__ int CeilDiv(int a, int b) { return (a-1)/b + 1; }
__device__ __host__ int CeilAlign(int a, int b) { return CeilDiv(a, b) * b; }

struct is_one
{
  __host__ __device__
  bool operator()(const int x)
  {
    return x == 1;
  }
};

// Some help kernel functions for accomplishing required tasks
__global__ void par_init(const char *text, int text_size, int *pos, int *lastpos)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx < text_size) {
		if(text[idx] == '\n')
			pos[idx] = lastpos[idx] = 0;
		else
			pos[idx] = lastpos[idx] = 1;
	}
}

__global__ void cal_position(int *pos, int *lastpos, int text_size, int i)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if(idx < text_size && idx-i <= 0) {
		if(lastpos[idx] != 0 && lastpos[idx] == lastpos[idx-1])
			pos[idx] += lastpos[idx-i];
	}	
}

__global__ void update(int *pos, int *lastpos, int text_size)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
			
	if(idx < text_size) 
		lastpos[idx] = pos[idx];
}

__global__ void set_char_mask(const char *text, int text_size, char ch, int *output)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(idx < text_size) {
		if(text[idx] == ch)
			output[idx] = 1;
		else
		output[idx] = 0;
	}
}


void CountPosition(const char *text, int *pos, int text_size)
{
	/* Write the letter positions in each word(separated by '\n') of *text into *pos */

	int blocksize = 512; // generally can't run more than 512 threads in a block
	int gridsize = text_size / blocksize + (text_size % blocksize == 0 ?0 : 1); // one thread per character
	int *lastpos; // array storing the last result for referencing 
	size_t arraysize = sizeof(int) * text_size;	
	hipMalloc((void **)&lastpos, arraysize);

	par_init<<<gridsize, blocksize>>>(text, text_size, pos, lastpos);
	// use this to synchronize kernel threads to prevent data hazards
	hipDeviceSynchronize();
	// look back to as far as 2^9 = 512 characters(longest word length = 500) behind the current one
	for(int i=0; i<10; i++) { 
		int lookback_dist = 1 << i;
		cal_position<<<gridsize, blocksize>>>(pos, lastpos, text_size, lookback_dist);
		hipDeviceSynchronize();
		// skip the last unnecessary update
		if(i == 9) break;
		update<<<gridsize, blocksize>>>(pos, lastpos, text_size);
		hipDeviceSynchronize();
	}
	hipFree(lastpos);
}

int ExtractHead(const int *pos, int *head, int text_size)
{
	/* Count */	
	// Use thrust functions to accomplish the tasks in this part.
	int *buffer;
	int nhead;
	hipMalloc(&buffer, sizeof(int)*text_size*2); // this is enough
	// wrap raw pointers into device pointers to use thrust functions
	// thrust::device_ptr<type> wrap_ptr_name(raw_ptr_name);
	thrust::device_ptr<const int> pos_d(pos);
	thrust::device_ptr<int> head_d(head), flag_d(buffer), cumsum_d(buffer+text_size);

	nhead = thrust::count(pos_d, pos_d+text_size, 1);

	thrust::equal_to<int> op_equal;
	// Use the flags as a mask to find the starting position of words
	thrust::fill(cumsum_d, cumsum_d+text_size, 1);  			
	thrust::transform(pos_d, pos_d+text_size, cumsum_d,flag_d,op_equal);
	thrust::sequence(cumsum_d, cumsum_d+text_size);
	thrust::copy_if(cumsum_d, cumsum_d+text_size, flag_d, head_d, is_one());  // write the result to head_d
	hipFree(buffer);
	return nhead;
}

void Part3(char *text, int *pos, int *head, int text_size, int n_head)
{	
	// find the occurences of the vowels
	int numA, numE, numI, numO, numU;
	int *buff;
	int blocksize = 512;
	int gridsize = text_size / blocksize + (text_size % blocksize == 0 ?0 : 1); // one thread per character
	size_t arraysize = sizeof(int) * text_size;	
	hipMalloc((void **)&buff, arraysize);

	thrust::device_ptr<int> buff_d(buff);
	
	set_char_mask<<<gridsize, blocksize>>>(text, text_size, 'A', buff);
	hipDeviceSynchronize();
	numA = thrust::count(buff_d, buff_d+text_size, 1);

	set_char_mask<<<gridsize, blocksize>>>(text, text_size, 'E', buff);
	hipDeviceSynchronize();
	numE = thrust::count(buff_d, buff_d+text_size, 1);

	set_char_mask<<<gridsize, blocksize>>>(text, text_size, 'I', buff);
	hipDeviceSynchronize();
	numI = thrust::count(buff_d, buff_d+text_size, 1);

	set_char_mask<<<gridsize, blocksize>>>(text, text_size, 'O', buff);
	hipDeviceSynchronize();
	numO = thrust::count(buff_d, buff_d+text_size, 1);

	set_char_mask<<<gridsize, blocksize>>>(text, text_size, 'U', buff);
	hipDeviceSynchronize();
	numU = thrust::count(buff_d, buff_d+text_size, 1);

	printf("frequency of A = %d, E = %d, I = %d, O = %d, U = %d\n", numA, numE, numI, numO, numU);
	hipFree(buff);
}
